#include <iostream>
#include <cstring>
#include <cstdlib>
#include <hip/hip_runtime.h>

#define DEFAULT_N 1024
#define NBLOCKS(n, block_size) ((n + block_size - 1) / block_size)

#define CUDA_CALL(call) \
  { \
    hipError_t err = call; \
    if (err != hipSuccess) { \
      std::cerr << __FILE__ << ":" << __LINE__ << ": CUDA call returned " \
        << err << ": " << hipGetErrorString(err) << "\n" \
        "    From " #call "\n"; \
      std::exit(1); \
    } \
  }

__global__ void dadd (const long N, const int niter, const double* a, const double* b, double* c)
{
  long i = blockIdx.x * blockDim.x + threadIdx.x;
  for (int n = 0; n < niter; n++) {
    if (i < N) c[i] = a[i] + b[i];
  }
}

void usage (char* arg0)
{
  std::cout << "Usage: " << arg0 << " N " << std::endl;
}

void init (const long N, const double scale, double* v)
{
#pragma omp parallel for shared(scale)
  for (long i = 0; i < N; i++) {
    v[i] = scale * (double)i;
  }
}

long verify (const long N, const double* h_a, const double* h_b, const double* h_c)
{
  double tol = 1e-7;
  long nerror = 0;
#pragma omp parallel for private(chk) reduction(+:nerror)
  for (long i = 0; i < N; i++) {
    double chk = h_c[i] - (h_a[i] + h_b[i]);
    if (chk >= tol) nerror += 1;
  }
  return nerror;
}

int main (int argc, char* argv[])
{
  // Parse N from argv[1]
  long N = DEFAULT_N;
  if (argc >= 2) {
    char* arg1 = argv[1];
    if (strcmp(arg1, "-h") == 0 || strcmp(arg1, "--help") == 0) {
      usage(argv[0]);
      return 0;
    }
    N = std::atol(arg1);
    if (N <= 0) {
      std::cerr << "ERROR: N must be positive\n";
      usage(argv[0]);
      return 2;
    }
  } // argc

  std::cout << "Using N = " << N << "\n";

  // Allocate host vectors
  double* h_A = (double*)malloc((size_t)N * sizeof(double));
  double* h_B = (double*)malloc((size_t)N * sizeof(double));
  double* h_C = (double*)malloc((size_t)N * sizeof(double));

  // Initialize host vectors
  init(N, 0.001, h_A);
  init(N, -0.001, h_B);

  // Allocate device vectors
  double* d_A; CUDA_CALL(hipMalloc(&d_A, (size_t)N * sizeof(double)));
  double* d_B; CUDA_CALL(hipMalloc(&d_B, (size_t)N * sizeof(double)));
  double* d_C; CUDA_CALL(hipMalloc(&d_C, (size_t)N * sizeof(double)));

  // Transfer input vectors to device
  CUDA_CALL(hipMemcpy(d_A, h_A, (size_t)N * sizeof(double), hipMemcpyHostToDevice));
  CUDA_CALL(hipMemcpy(d_B, h_B, (size_t)N * sizeof(double), hipMemcpyHostToDevice));

  // Perform vector addition using CUDA kernel
  int niter = 100;
  dadd<<<NBLOCKS(N, 1024), 1024>>>(N, niter, d_A, d_B, d_C);
  CUDA_CALL(hipGetLastError());
  CUDA_CALL(hipDeviceSynchronize());

  // Transfer result vector to host
  CUDA_CALL(hipMemcpy(h_C, d_C, (size_t)N * sizeof(double), hipMemcpyDeviceToHost));

  // Verify results
  long nerr = verify(N, h_A, h_B, h_C);
  if (nerr > 0) {
    std::cerr << "ERROR: " << nerr << " elements differ\n";
    return 1;
  }

  // Clean up device vectors
  CUDA_CALL(hipFree(d_A));
  CUDA_CALL(hipFree(d_B));
  CUDA_CALL(hipFree(d_C));

  // Clean up host vectors
  free(h_A);
  free(h_B);
  free(h_C);

  return 0;
}
